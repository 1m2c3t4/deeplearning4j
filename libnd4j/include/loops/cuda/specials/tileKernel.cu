#include "hip/hip_runtime.h"
/*******************************************************************************
 * Copyright (c) 2015-2018 Skymind, Inc.
 *
 * This program and the accompanying materials are made available under the
 * terms of the Apache License, Version 2.0 which is available at
 * https://www.apache.org/licenses/LICENSE-2.0.
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS, WITHOUT
 * WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied. See the
 * License for the specific language governing permissions and limitations
 * under the License.
 *
 * SPDX-License-Identifier: Apache-2.0
 ******************************************************************************/

//
// @author GS <sgazeos@gmail.com>, created on 16.01.2019
//

#include <loops/special_kernels.h>

namespace nd4j {

////////////////////////////////////////////////////////////////////////
    template<typename T>
    static __global__ void
    tileKernel(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape,
               Nd4jLong resultLength) {
////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
//        Original code to transform in cuda-based
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;
        //const auto resultLength = shape::length(outputShape);
        if (shape::order(outputShape) == 'c') {           //  ews == 1 always here
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = shape::subArrayIndex(outputShape, inputShape, i);
                *(reinterpret_cast<T *>(outputBuffer) + i) = *(reinterpret_cast<T const *>(inputBuffer) + yOffset);
            }
//            for(Nd4jLong i=0;  i<resultLen; ++i) {
//                auto yOffset = shape::subArrayIndex(newShapeInfo, _shapeInfo, i);
//                BUILD_SINGLE_SELECTOR(xType, this->template templatedAssign, (newBuff, i, this->_buffer, yOffset), LIBND4J_TYPES);
//
//            }
        } else {
//
            //auto inputLength = shape::lenght(inputShape);
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto xOffset = shape::getIndexOffset(i, outputShape, resultLength);
                auto yOffset = shape::subArrayIndex(outputShape, inputShape, i);
                *(reinterpret_cast<T *>(outputBuffer) + xOffset) = *(reinterpret_cast<T const *>(inputBuffer) +
                                                                     yOffset);
//                BUILD_SINGLE_SELECTOR(xType, this->template templatedAssign, (newBuff, xOffset, this->_buffer, yOffset), LIBND4J_TYPES);
            }
        }

    }

    BUILD_SINGLE_TEMPLATE(template __global__ void tileKernel,
                          (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength),
                          LIBND4J_TYPES);

    template<typename T>
    void tileKernelH(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape,
                     Nd4jLong resultLength, hipStream_t stream) {
        dim3 launchDims(256, 512, 8192);
        tileKernel<T> << < launchDims.x, launchDims.y, launchDims.z, stream >> >
                                                                     (inputBuffer, inputShape, outputBuffer, outputShape, resultLength);
    }

    BUILD_SINGLE_TEMPLATE(template void tileKernelH,
                          (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, hipStream_t stream),
                          LIBND4J_TYPES);

    template<typename X, typename Y>
    static __global__ void
    tileKernelDouble(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape,
                     Nd4jLong resultLength, Nd4jLong ews) {
        char ordering = shape::order(outputShape);
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        int totalThreads = gridDim.x * blockDim.x;

        if (ordering == 'c' && ews == 1) {           //  ews == 1 always here
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = shape::subArrayIndex(outputShape, inputShape, i);
                *(reinterpret_cast<X *>(outputBuffer) + i) = static_cast<X>(*(reinterpret_cast<Y const *>(inputBuffer) +
                                                                              yOffset));
            }
        } else if (ordering == 'c' && ews > 1) {
            for (int i = tid; i < resultLength; i += totalThreads) {
                auto yOffset = shape::subArrayIndex(outputShape, inputShape, i);
                *(reinterpret_cast<X *>(outputBuffer) + i * ews) = static_cast<X>(*(
                        reinterpret_cast<Y const *>(inputBuffer) + yOffset));
            }
        } else {

            for (int i = tid; i < resultLength; i += totalThreads) {

                auto xOffset = shape::getIndexOffset(i, outputShape, resultLength);
                auto yOffset = shape::subArrayIndex(outputShape, inputShape, i);
                *(reinterpret_cast<X *>(outputBuffer) + xOffset) = static_cast<X>(*(
                        reinterpret_cast<Y const *>(inputBuffer) + yOffset));
            }
        }
    }

    BUILD_DOUBLE_TEMPLATE(template __global__ void tileKernelDouble,
                          (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, Nd4jLong ews),
                          LIBND4J_TYPES, LIBND4J_TYPES);

    template<typename X, typename Y>
    void tileKernelHH(void const *inputBuffer, Nd4jLong *inputShape, void *outputBuffer, Nd4jLong *outputShape,
                      Nd4jLong resultLength, Nd4jLong ews, hipStream_t stream) {
        dim3 launchDims(256, 512, 8192);
        tileKernelDouble<X, Y> << < launchDims.x, launchDims.y, launchDims.z, stream >> >
                                                                              (inputBuffer, inputShape, outputBuffer, outputShape, resultLength, ews);
    }

    BUILD_DOUBLE_TEMPLATE(template void tileKernelHH,
                          (void const* inputBuffer, Nd4jLong* inputShape, void* outputBuffer, Nd4jLong* outputShape, Nd4jLong resultLength, Nd4jLong ews, hipStream_t stream),
                          LIBND4J_TYPES, LIBND4J_TYPES);



    template <typename Lambda>
    __global__ void runLambda(double *input, double *output, Nd4jLong length, Lambda lambda) {
        auto tid = blockIdx.x * blockDim.x + threadIdx.x;
        float16 f(1.0f);
        for (Nd4jLong e = tid; e < length; e += gridDim.x * blockDim.x) {
            output[e] = lambda(input[e]) + (double) f;
        }
    }

    void launcher(hipStream_t *stream, double *input, double *output, Nd4jLong length) {
        auto f = [] __device__ (double x) -> double {
            return x + 1.;
        };

        runLambda<<<128, 128, 128, *stream>>>(input, output, length, f);
    }
}